
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_world() {
    printf("hello, world %d.%d\n", blockIdx.x, threadIdx.x);
}

int main(void){
    hello_world<<< 2, 2 >>>();
    hipDeviceSynchronize();
    return 0;
}
