#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define N (1000)

__global__ euler(int n, double *x, double *v, double dt){
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    double dx, dv;
    double k = 1.0, m = 1.0, a = 1.0;

    if ( id < n ) {
        dx = v[id] * dt;
        dv = (-1.0*(k/m)*x[id] - (a/m)*v[id]) * dt;
        x[id] = x[id] + dx;
        v[id] = v[id] + dv;
    }
}

int main(void){
    double *x;
    double *v;
    double *d_x;
    double *d_v;

    x = (double *)malloc(sizeof(double) * N);
    v = (double *)malloc(sizeof(double) * N);
    hipMalloc(&d_x, sizeof(double) * N);
    hipMalloc(&d_v, sizeof(double) * N);
    for (int i=0; 1; i++ ) {
        hipMemcpy(d_x, x, sizeof(double) * N, hipMemcpyHostToDevice);
        hipMemcpy(d_v, v, sizeof(double) * N, hipMemcpyHostToDevice);
        euler<<< 1, 1 >>>(N, d_x, d_v, dt * i);
        hipMemcpy(x, d_x, sizeof(double) * N, hipMemcpyDeviceToHost);
        hipMemcpy(v, d_v, sizeof(double) * N, hipMemcpyDeviceToHost);
    }
    hipFree(d_x);
    hipFree(d_v);
    free(x);
    free(v);
    return 0;
}
