#include "hip/hip_runtime.h"
#include <stdio.h>

#define HEIGHT 9
#define WIDTH 9

__global__ void kernel(int* target){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if ( id < (HIGHT*WIDTH) ) {
        target[blockIdx.x*blockDim.x + threadIdx.x] = (blockIdx.x + 1) * (threadIdx.x + 1);
    }
}

int main(){
    int array[HEIGHT*WIDTH];
    int *device_array;

    hipMalloc((void**)&device_array, sizeof(int) * HEIGHT*WIDTH);
    kernel<<<HEIGHT, WIDTH>>>(device_array);
    hipMemcpy(array, device_array, sizeof(int) * HEIGHT*WIDTH, hipMemcpyDeviceToHost);
    hipFree(device_array);

    for(int y=0; y<HEIGHT; y++){
        for(int x=0; x<WIDTH; x++){
            printf("%d\t", array[y*WIDTH + x]);
        }
        printf("\n");
    }
    return 0;
}
